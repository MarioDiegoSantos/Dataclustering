#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <sstream>
#include "gpu_cgrasp.h"


/********* XINSHEYANG2 ************/

__device__ float d_XINSHEYANG2(float *x, int n) {
	float sum1 = 0;
	for(int i = 0; i < n; i++) {
		sum1 += fabs(x[i]);
	}
	
	float sum2 = 0;
	for(int i = 0; i < n; i++) {
		sum2 += sinf(powf(x[i],2));
	}

	return sum1 * expf(-sum2);
}

float h_XINSHEYANG2(float *x, int n) {
	float sum1 = 0;
	for(int i = 0; i < n; i++) {
		sum1 += fabs(x[i]);
	}
	
	float sum2 = 0;
	for(int i = 0; i < n; i++) {
		sum2 += sinf(powf(x[i],2));
	}

	return sum1 * expf(-sum2);
}

__device__ fptr_t d_p_XINSHEYANG2 = d_XINSHEYANG2;

/********* PINTER ************/

__device__ float d_PINTER(float *x, int n) {
	float sum1 = 0, sum2 = 0, sum3 = 0;

	for(int i = 0; i < n; i++) {
		float A, B;
		if(i==0) {
			A = x[n-1] * sinf(x[i]) + sinf(x[i+1]);
			B = powf(x[n-1], 2) - 2*x[i] + 3*x[i+1] - cosf(x[i]) + 1;
		} else if(i==n-1) {
			A = x[i-1] * sinf(x[i]) + sinf(x[0]);
			B = powf(x[i-1], 2) - 2*x[i] + 3*x[0] - cosf(x[i]) + 1;
		} else {
			A = x[i-1] * sinf(x[i]) + sinf(x[i+1]);
			B = powf(x[i-1], 2) - 2*x[i] + 3*x[i+1] - cosf(x[i]) + 1;
		}
		
		sum1 += (i+1) * powf(x[i],2);
		sum2 += 20 * (i+1) * powf(sinf(A),2);
		sum3 += (i+1) * log10f(1+(i+1)*powf(B,2));
	}

	return sum1 + sum2 + sum3;
}

float h_PINTER(float *x, int n) {
	float sum1 = 0, sum2 = 0, sum3 = 0;

	for(int i = 0; i < n; i++) {
		float A, B;
		if(i==0) {
			A = x[n-1] * sinf(x[i]) + sinf(x[i+1]);
			B = powf(x[n-1], 2) - 2*x[i] + 3*x[i+1] - cosf(x[i]) + 1;
		} else if(i==n-1) {
			A = x[i-1] * sinf(x[i]) + sinf(x[0]);
			B = powf(x[i-1], 2) - 2*x[i] + 3*x[0] - cosf(x[i]) + 1;
		} else {
			A = x[i-1] * sinf(x[i]) + sinf(x[i+1]);
			B = powf(x[i-1], 2) - 2*x[i] + 3*x[i+1] - cosf(x[i]) + 1;
		}
		
		sum1 += (i+1) * powf(x[i],2);
		sum2 += 20 * (i+1) * powf(sinf(A),2);
		sum3 += (i+1) * log10f(1+(i+1)*powf(B,2));
	}

	return sum1 + sum2 + sum3;
}

__device__ fptr_t d_p_PINTER = d_PINTER;

/********* PATHOLOGICAL ************/

__device__ float d_PATHOLOGICAL(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n-1; i++) {
		float num = powf(sinf(sqrtf(100* powf(x[i],2) + powf(x[i+1],2))),2) - 0.5;
		float den = 1 + 0.001*powf(powf(x[i],2)-2*x[i]*x[i+1]+powf(x[i+1],2),2);
		sum += 0.5 + num/den;
	}
	return sum;
}

float h_PATHOLOGICAL(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n-1; i++) {
		float num = powf(sinf(sqrtf(100* powf(x[i],2) + powf(x[i+1],2))),2) - 0.5;
		float den = 1 + 0.001*powf(powf(x[i],2)-2*x[i]*x[i+1]+powf(x[i+1],2),2);
		sum += 0.5 + num/den;
	}
	return sum;
}

__device__ fptr_t d_p_PATHOLOGICAL = d_PATHOLOGICAL;

/********* ZAKHAROV ************/

__device__ float d_ZAKHAROV(float *x, int n) {
	float sum1 = 0;
	for(int i = 0; i < n; i++) {
		sum1 += powf(x[i],2);
	}

	float sum2 = 0;
	float sum3 = 0;
	for(int i = 0; i < n; i++) {
		sum2 += (i+1)*x[i];
	}
	sum2 = 1/2 * sum2;
	sum3 = powf(sum2,4);
	sum2 = powf(sum2,2);

	return sum1+sum2+sum3;
}

float h_ZAKHAROV(float *x, int n) {
	float sum1 = 0;
	for(int i = 0; i < n; i++) {
		sum1 += powf(x[i],2);
	}

	float sum2 = 0;
	float sum3 = 0;
	for(int i = 0; i < n; i++) {
		sum2 += (i+1)*x[i];
	}
	sum2 = 1/2 * sum2;
	sum3 = powf(sum2,4);
	sum2 = powf(sum2,2);

	return sum1+sum2+sum3;
}

__device__ fptr_t d_p_ZAKHAROV = d_ZAKHAROV;


/********* QING ************/

__device__ float d_QING(float *x, int n) {
	float sum = 0;
	for(int i = 1; i <= n; i++) {
		sum += powf(powf(x[i-1], 2) - i,2);
	}
	return sum;
}

float h_QING(float *x, int n) {
	float sum = 0;
	for(int i = 1; i <= n; i++) {
		sum += powf(powf(x[i-1], 2) - i,2);
	}
	return sum;
}

__device__ fptr_t d_p_QING = d_QING;

/********* POWELLSINGULAR ************/

__device__ float d_POWELLSINGULAR(float *x, int n) {
	float sum = 0;
	for(int i = 1; i <= n/4; i++) {		
		sum += powf(x[4*i-4] + 10 * x[4*i-3],2) + 
				5 * powf(x[4*i-2] - x[4*i-1],2) +
				powf(x[4*i-3] - x[4*i-2],4) +
				10*powf(x[4*i-4] - x[4*i-1],4);
	}
	return sum;
}

float h_POWELLSINGULAR(float *x, int n) {
	float sum = 0;
	for(int i = 1; i <= n/4; i++) {		
		sum += powf(x[4*i-4] + 10 * x[4*i-3],2) + 
				5 * powf(x[4*i-2] - x[4*i-1],2) +
				powf(x[4*i-3] - x[4*i-2],4) +
				10*powf(x[4*i-4] - x[4*i-1],4);
	}
	return sum;
}

__device__ fptr_t d_p_POWELLSINGULAR = d_POWELLSINGULAR;

/********* POWELLSUM ************/

__device__ float d_POWELLSUM(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += powf(fabs(x[i]), i+2);
	}
	return sum;
}

float h_POWELLSUM(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += powf(fabs(x[i]), i+2);
	}
	return sum;
}

__device__ fptr_t d_p_POWELLSUM = d_POWELLSUM;

/********* MISHRA2 ************/

__device__ float d_MISHRA2(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n-1; i++) {
		sum += x[i] + x[i+1];
	}
	sum = 0.5 * sum;
	return powf(1 + n - sum, n - sum);
}

float h_MISHRA2(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n-1; i++) {
		sum += x[i] + x[i+1];
	}
	sum = 0.5 * sum;

	return powf(1 + n - sum, n - sum);
}

__device__ fptr_t d_p_MISHRA2 = d_MISHRA2;


/********* MISHRA1 ************/

__device__ float d_MISHRA1(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n-1; i++) {
		sum += x[i];
	}
	

	return powf(1 + n - sum, n - sum);
}

float h_MISHRA1(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n-1; i++) {
		sum += x[i];
	}
	

	return powf(1 + n - sum, n - sum);
}

__device__ fptr_t d_p_MISHRA1 = d_MISHRA1;

/********* EXPONENTIAL ************/

__device__ float d_EXPONENTIAL(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += powf(x[i],2);
	}
	sum = -0.5 * sum;

	return -expf(sum);
}

float h_EXPONENTIAL(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += powf(x[i],2);
	}
	sum = -0.5 * sum;

	return -expf(sum);
}

__device__ fptr_t d_p_EXPONENTIAL = d_EXPONENTIAL;

/********* BROWN ************/

__device__ float d_BROWN(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n-1; i++) {
		sum += powf(powf(x[i+1],2), powf(x[i],2) + 1) + powf(powf(x[i],2), powf(x[i+1],2) + 1);
	}	
	return sum;
}

float h_BROWN(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n-1; i++) {
		sum += powf(powf(x[i+1],2), powf(x[i],2) + 1) + powf(powf(x[i],2), powf(x[i+1],2) + 1);
	}	
	return sum;
}

__device__ fptr_t d_p_BROWN = d_BROWN;

/********* ALPINE1 ************/

__device__ float d_ALPINE1(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += fabs(x[i] * sinf(x[i]) + 0.1 * x[i]);
	}	
	return sum;
}

float h_ALPINE1(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += fabs(x[i] * sinf(x[i]) + 0.1 * x[i]);
	}	
	return sum;
}

__device__ fptr_t d_p_ALPINE1 = d_ALPINE1;



/********* CHUNG ************/

__device__ float d_CHUNG(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += powf(x[i], 2);
	}	
	return powf(sum, 2);
}

float h_CHUNG(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += powf(x[i], 2);
	}	
	return powf(sum, 2);
}

__device__ fptr_t d_p_CHUNG = d_CHUNG;


/********* SPHERE ************/

__device__ float d_SPHERE(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += powf(x[i], 2);
	}	
	return sum;
}

float h_SPHERE(float *x, int n) {
	float sum = 0;
	for(int i = 0; i < n; i++) {
		sum += powf(x[i], 2);
	}	
	return sum;
}

__device__ fptr_t d_p_SPHERE = d_SPHERE;


/********* ACKLEY1 ************/

__device__ float d_ACKLEY1(float *x, int n) {
	float sum1 = 0;
	float sum2 = 0;

	for(int i = 0; i < n; i++) {
		sum1 += powf(x[i], 2);
		sum2 += cosf(2*M_PI*x[i]);
	}

	return (-20 * expf(-0.2 * sqrtf(1.0/n * sum1)) - expf(1.0/n * sum2) + 20 + expf(1));
}

float h_ACKLEY1(float *x, int n) {
	float sum1 = 0;
	float sum2 = 0;

	for(int i = 0; i < n; i++) {
		sum1 += powf(x[i], 2);
		sum2 += cosf(2*M_PI*x[i]);
	}

	return (-20 * expf(-0.2 * sqrtf(1.0/n * sum1)) - expf(1.0/n * sum2) + 20 + expf(1));
}

__device__ fptr_t d_p_ACKLEY1 = d_ACKLEY1;


/********* RASTRIGIN ************/

__device__ float d_RASTRIGIN(float *x, int n) {
	float sum = 10 * n;
	for(int i=0; i < n; i++) {
		sum += powf(x[i],2) - 10 * cosf(2 * M_PI * x[i]);
	}

	return sum;
}

float h_RASTRIGIN(float *x, int n) {
	float sum = 10 * n;
	for(int i=0; i < n; i++) {
		sum += powf(x[i],2) - 10 * cosf(2 * M_PI * x[i]);
	}

	return sum;
}

__device__ fptr_t d_p_RASTRIGIN = d_RASTRIGIN;

/********* DIXON ************/

__device__ float d_DIXON(float *x, int n) {
	float sum = powf(x[0] - 1, 2);
	for(int i=1; i < n; i++) {
		sum += (i+1) * powf( 2*powf(x[i],2) - x[i-1], 2);
	}

	return sum;
}

float h_DIXON(float *x, int n) {
	float sum = powf(x[0] - 1, 2);
	for(int i=1; i < n; i++) {
		sum += (i+1) * powf( 2*powf(x[i],2) - x[i-1], 2);
	}

	return sum;
}

__device__ fptr_t d_p_DIXON = d_DIXON;

/********* ROSENBROCK ************/

__device__ float d_ROSENBROCK(float *x, int n) {
	float sum = 0;
 
	for(int i=0; i < n-1; i++) {
		sum += 100*powf(x[i+1] - powf(x[i],2),2) + powf(x[i] - 1, 2);
	}

	return sum;	
}

float h_ROSENBROCK(float *x, int n) {
	float sum = 0;

	for(int i=0; i < n-1; i++) {
		sum += 100*powf(x[i+1] - powf(x[i],2),2) + powf(x[i] - 1, 2);
	}

	return sum;	
}

__device__ fptr_t d_p_ROSENBROCK = d_ROSENBROCK;

/********* STYBLINSK ************/

__device__ float d_STYBLINSK(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n; i++) {
		sum += powf(x[i],4) - 16*powf(x[i],2) + 5*x[i];
	}

	return sum/2;	
}

float h_STYBLINSK(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n; i++) {
		sum += powf(x[i],4) - 16*powf(x[i],2) + 5*x[i];
	}

	return sum/2;	
}

__device__ fptr_t d_p_STYBLINSK = d_STYBLINSK;

/********* SCHWEFEL ************/

__device__ float d_SCHWEFEL(float *x, int n) {
	float sum = 418.9829 * n;
	for(int i=0; i < n; i++) {
		sum += x[i] * sinf(sqrtf(fabs(x[i])));
	}
	if (sum < 0) sum = 0;
	return sum;	
}

float h_SCHWEFEL(float *x, int n) {
	float sum = 418.9829 * n;
	for(int i=0; i < n; i++) {
		sum += x[i] * sinf(sqrtf(fabs(x[i])));
	}
	
	if (sum < 0) sum = 0;
	return sum;	
}

__device__ fptr_t d_p_SCHWEFEL = d_SCHWEFEL;

/********* SALOMON ************/

__device__ float d_SALOMON(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n; i++) {
		sum += powf(x[i],2);
	}
	sum = sqrtf(sum);
	return (1 - cosf(2.0f*M_PI*sum) + 0.1f * sum);
}

float h_SALOMON(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n; i++) {
		sum += powf(x[i],2);
	}
	sum = sqrtf(sum);
	return (1 - cosf(2.0f*M_PI*sum) + 0.1f * sum);
}

__device__ fptr_t d_p_SALOMON = d_SALOMON;

/********* GRIEWANK ************/

__device__ float d_GRIEWANK(float *x, int n) {
	float sum1 = 0;
	for(int i=0; i < n; i++) {
		sum1 += pow(x[i],2);
	}
	sum1 = sum1/4000;

	float sum2 = 1;
	for(int i=0; i < n; i++) {
		sum2 *= cosf(x[i]/sqrtf(i+1));
	}

	return (1 + sum1 - sum2);	
}

float h_GRIEWANK(float *x, int n) {
	float sum1 = 0;
	for(int i=0; i < n; i++) {
		sum1 += pow(x[i],2);
	}
	sum1 = sum1/4000;

	float sum2 = 1;
	for(int i=0; i < n; i++) {
		sum2 *= cosf(x[i]/sqrtf(i+1));
	}

	return (1 + sum1 - sum2);	
}

__device__ fptr_t d_p_GRIEWANK = d_GRIEWANK;

/********* SCHAFER_F6 ************/

__device__ float d_SCHAFER_F6(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n-1; i++) {
		sum += 0.5f+((powf(sinf(sqrtf(powf(x[i], 2) + powf(x[i+1], 2))),2)-0.5f)/powf(1 + 0.001f*(powf(x[i], 2) + 
					powf(x[i+1], 2)), 2));
	}
	return sum;
}

float h_SCHAFER_F6(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n-1; i++) {
		sum += 0.5f+((powf(sinf(sqrtf(powf(x[i], 2) + powf(x[i+1], 2))),2)-0.5f)/powf(1 + 0.001f*(powf(x[i], 2) + 
					powf(x[i+1], 2)), 2));
	}
	return sum;
}

__device__ fptr_t d_p_SCHAFER_F6 = d_SCHAFER_F6;


/********* F25aF28 ************/

__device__ float d_F25aF28(float *x, int n) {
	float sum1 = 0, sum2 = 0;
	for(int i=0; i < n; i++) {
		sum1 += powf(x[i], 2)/powf(2, i);
		if(i<n-1)
			sum2 += powf(x[i+1]*x[i], 2)/powf(2, i+1);
	}
	return sum1+sum2;
}

float h_F25aF28(float *x, int n) {
	float sum1 = 0, sum2 = 0;
	for(int i=0; i < n; i++) {
		sum1 += powf(x[i], 2)/powf(2, i);
		if(i<n-1)
			sum2 += powf(x[i+1]*x[i], 2)/powf(2, i+1);
	}
	return sum1+sum2;
}

__device__ fptr_t d_p_F25aF28 = d_F25aF28;

/********* LEVY ************/

__device__ float d_LEVY(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n-1; i++) {
		sum += powf(x[i]-1,2) * (1+10*powf(sinf(M_PI*x[i]),2));
	}
	return powf(sinf(M_PI * x[0]),2) + sum + powf(x[n-1]-1,2) * (10*powf(sinf(M_PI*x[n-1]),2));
}

float h_LEVY(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n-1; i++) {
		sum += powf(x[i]-1,2) * (1+10*powf(sinf(M_PI*x[i]),2));
	}
	return powf(sinf(M_PI * x[0]),2) + sum + powf(x[n-1]-1,2) * (10*powf(sinf(M_PI*x[n-1]),2));
}

__device__ fptr_t d_p_LEVY = d_LEVY;

/********* PICCIONI ************/

__device__ float d_PICCIONI(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n-1; i++) {
		sum += powf(x[i]-1,2) * (1+10*powf(sinf(M_PI*x[i+1]),2));
	}
	return 10*powf(sinf(M_PI * x[0]),2) + sum + powf(x[n-1]-1,2);
}

float h_PICCIONI(float *x, int n) {
	float sum = 0;
	for(int i=0; i < n-1; i++) {
		sum += powf(x[i]-1,2) * (1+10*powf(sinf(M_PI*x[i+1]),2));
	}
	return 10*powf(sinf(M_PI * x[0]),2) + sum + powf(x[n-1]-1,2);
}

__device__ fptr_t d_p_PICCIONI = d_PICCIONI;

/********* CLUSTER MARIO *********/ // Como acessar points dessa funcao [BRUNO]
__device__ float* g_d_points; //esse é o global device
//__constant__ int g_d_dim; 
	//__constant__ int g_d_npoints; 

float* points;
const float my_pi = 3.14;


__device__ float d_CLUSTER(float *x, int n) {
	float sum = 0;
	return g_d_points[0] ; //[BRUNO] ERRO está aqui!
}

float h_CLUSTER(float *x, int n) {
	float sum = 0;

	return points[0];
}

__device__ fptr_t d_p_CLUSTER = d_CLUSTER;



int main(int argc, char **argv){
	fptr_t d_obj_f, h_obj_f;
	int n = 25;
	double cutoff_time = 60;
	float hs = 0.5, he = 0.0001, ep = 1/powf(2,13);
	int gpu = 1;
	int max_points = 128;
	int seed = time(NULL);

	Results res1, res2, res3, res4;



	int n_points;
	int dim;


	std::string s;
	
	std::getline(std::cin, s);
	std::stringstream st(s);
	st >> n_points >> dim;
	points = new float[n_points*dim];
	n = dim;

	float CLUSTER_L[n], CLUSTER_U[n];

	for(int i=0;i<n;++i) {
		CLUSTER_L[i] = 1000;
		CLUSTER_U[i] = -1000;
	}

	for(int i=0;i<n_points;++i) {
		
		std::getline(std::cin, s);
		std::stringstream st(s);
		for(int d=0; d<dim; ++d) {
			st >> points[4*i + d];

			if(CLUSTER_L[d] > points[4*i + d])
				CLUSTER_L[d] = points[4*i + d]; //Min da dim
			if(CLUSTER_U[d] < points[4*i + d])
				CLUSTER_U[d] = points[4*i + d]; //Max da dim

		}



	}
        float *d_points; // 


        hipMalloc( &d_points, n_points * dim * sizeof(float)); //aloco e copio
        hipMemcpy(d_points, points, n_points * dim * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpyToSymbol(HIP_SYMBOL(g_d_points), &d_points, sizeof(d_points));



       //   hipMemcpyToSymbol(HIP_SYMBOL(g_d_dim), &dim, sizeof(int));
       // hipMemcpyToSymbol(HIP_SYMBOL(g_d_npoints), &n_points, sizeof(int));

	/*for(int i=0;i<n_points;i++){
		for(int d=0;d<dim;d++)
			std::cout << points[i][d] << " ";
		std::cout << std::endl;
	}*/

        //gpuErrchk(hipMalloc(&d_points, n_points* dim * sizeof(float)));



	
	/*************** CLUSTER ***********************/


	hipMemcpyFromSymbol(&d_obj_f, HIP_SYMBOL(d_p_CLUSTER), sizeof (fptr_t));
	h_obj_f = h_CLUSTER;

	// // float POWELLSINGULAR_L[n], POWELLSINGULAR_U[n];
	// // for (int i = 0; i < n; i++) {
	// // 	POWELLSINGULAR_L[i] = -4;
	// // 	POWELLSINGULAR_U[i] = 5;
	// // }


	//res1 = gpu_cgrasp(n, CLUSTER_L, CLUSTER_U, points, n_points, dim, 0, seed, ep, hs, he, cutoff_time,
	 //			   	  max_points, 0, h_obj_f, d_obj_f);

	printf("total time: %lf\n", res1.time);
	res2 = gpu_cgrasp(n, CLUSTER_L, CLUSTER_U, points, n_points, dim, 0.0, seed, ep, hs, he, cutoff_time,
				   	  max_points, 1, h_obj_f, d_obj_f);

	//res3 = pcgrasp(n, POWELLSINGULAR_L, POWELLSINGULAR_U, points, n_points, dim, 0.0, seed, ep, hs, he, cutoff_time,
	 //			   	  max_points, 0, h_obj_f, d_obj_f);

	// res4 = pcgrasp(n, POWELLSINGULAR_L, POWELLSINGULAR_U, 0.0, seed, ep, hs, he, cutoff_time,
	//  			   	  max_points, 1, h_obj_f, d_obj_f);

	printf("\n\nInstance: CLUSTER\n");
	printf("total time: %lf\n", res1.time);
	printf("total time (gpu): %lf\n", res2.time);
	printf("total time (par): %lf\n", res3.time);
	printf("total time (gpar): %lf\n", res4.time);
	printf("total evaluations: %ld\n", res1.evaluations);
	printf("total evaluations (gpu): %ld\n", res2.evaluations);
	printf("total evaluations (par): %ld\n", res3.evaluations);
	printf("total evaluations (gpar): %ld\n", res4.evaluations);
	printf("error: %lf\n", fabs(res1.best));
	printf("error (gpu): %lf\n", fabs(res2.best));
	printf("error (par): %lf\n", fabs(res3.best));
	printf("error (gpar): %lf\n", fabs(res4.best));



	return 0;
}
